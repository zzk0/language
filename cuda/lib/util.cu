#include "hip/hip_runtime.h"
#include "util.h"
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>

namespace {

/**
用于比较两个矩阵是否相同。
列优先，(i, j) 对应位置为 i + j * ldx (ldx = 行的个数)
*/
__global__ void EqualCheckCUDAimpl(float *A, float *B, int numel, int *status) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx < numel) {
    if (abs(A[tx] - B[tx]) > 1e-3) {
      printf("not equal: %f %f\n", A[tx], B[tx]);
      *status += 1;
    }
  }
}

}

bool EqualCheckCUDA(float *dev_a, float *dev_b, int numel, bool verbose) {
  const auto& t = GetGridAndBlock(numel);
  const auto& grid = std::get<0>(t);
  const auto& block = std::get<1>(t);
  int status = 0;
  int *dev_status;
  hipMalloc(&dev_status, sizeof(int));
  hipMemcpy(dev_status, &status, sizeof(int), hipMemcpyHostToDevice);
  EqualCheckCUDAimpl<<<grid, block>>>(dev_a, dev_b, numel, dev_status);
  hipMemcpy(&status, dev_status, sizeof(int), hipMemcpyDeviceToHost);
  return (status == 0);
}
