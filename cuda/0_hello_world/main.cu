
#include <hip/hip_runtime.h>
#include "iostream"
#include "stdio.h"

__device__ int devData;
__global__ void counter() {
  printf("%i\n", devData);
  devData += 1;
}

__global__ void hello_world() {
//  std::cout << "Hello Cuda" << std::endl;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  printf("%d %d\n", i, j);
}

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  std::cout << "Device Count: " << deviceCount << std::endl;

  int x = hipDeviceEnablePeerAccess(1, 0);
  std::cout << (x == hipSuccess) << std::endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  std::cout << prop.name << std::endl;
  std::cout << prop.multiProcessorCount << std::endl;
  std::cout << prop.sharedMemPerBlock << std::endl;
  std::cout << prop.maxThreadsPerBlock << std::endl;
  std::cout << prop.maxThreadsPerMultiProcessor << std::endl;
  std::cout << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << std::endl;

  hello_world<<<1, 5>>>();
  hipDeviceSynchronize();

  return 0;
}
